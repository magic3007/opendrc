#include "hip/hip_runtime.h"
#include <odrc/algorithm/space-check.hpp>

#include <cassert>

#include <iostream>
#include <stack>
#include <stdexcept>
#include <string>
#include <type_traits>
#include <unordered_map>
#include <variant>

#include <hip/hip_runtime.h>

#include <odrc/core/cell.hpp>

namespace odrc {

using coord    = odrc::core::coord;
using polygon  = odrc::core::polygon;
using cell_ref = odrc::core::cell_ref;
using h_edge   = odrc::core::h_edge;
using v_edge   = odrc::core::v_edge;

struct check_result {
  int  e11x;
  int  e11y;
  int  e12x;
  int  e12y;
  int  e21x;
  int  e21y;
  int  e22x;
  int  e22y;
  bool is_violation = false;
};

__global__ void space_check_kernel(coord*        coords1,
                                   coord*        coords2,
                                   int           size1,
                                   int           size2,
                                   int           start,
                                   int           threshold,
                                   check_result* results) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= size1 * size2 / 2) {
    return;
  }

  int p1 = tid / size2;
  int p2 = tid % size2;

  check_result& res =
      start == 0 ? results[tid] : results[tid + size1 * size2 / 4];

  res.e11x = coords1[p1].x;
  res.e11y = coords1[p1].y;
  res.e12x = coords1[p1 + 1].x;
  res.e12y = coords1[p1 + 1].y;
  res.e21x = coords2[p2].x;
  res.e21y = coords2[p2].y;
  res.e22x = coords2[p2 + 1].x;
  res.e22y = coords2[p2 + 1].y;

  // space check
  if (res.e11x == res.e12x) {  // vertical
    if (res.e11x < res.e21x) {
      // e11 e22
      // e12 e21
      bool is_outside_to_outside = res.e11y > res.e12y and res.e21y < res.e22y;
      bool is_too_close          = res.e21x - res.e11x < threshold;
      bool is_projection_overlap = res.e11y < res.e21y and res.e22y < res.e12y;
      res.is_violation =
          is_outside_to_outside and is_too_close and is_projection_overlap;
      if (res.is_violation) {
        printf("T[%d]: (%d, %d), (%d, %d), (%d, %d), (%d, %d)\n", tid, res.e11x,
               res.e11y, res.e12x, res.e12y, res.e21x, res.e21y, res.e22x,
               res.e22y);
      }

    } else {
      // e21 e12
      // e22 e11
      bool is_outside_to_outside = res.e21y > res.e22y and res.e11y < res.e21y;
      bool is_too_close          = res.e11x - res.e21x < threshold;
      bool is_projection_overlap = res.e21y < res.e11y and res.e12y < res.e22y;
      res.is_violation =
          is_outside_to_outside and is_too_close and is_projection_overlap;
    }
  } else {  // horizontal
    if (res.e11y < res.e22y) {
      // e22 e21
      // e11 e12
      bool is_outside_to_outside = res.e11x < res.e12x and res.e21x > res.e22x;
      bool is_too_close          = res.e21y - res.e11y < threshold;
      bool is_projection_overlap = res.e21x < res.e11x and res.e12x < res.e22x;
      res.is_violation =
          is_outside_to_outside and is_too_close and is_projection_overlap;
    } else {
      // e12 e11
      // e21 e22
      bool is_outside_to_outside = res.e21x < res.e22x and res.e11x > res.e12x;
      bool is_too_close          = res.e11y - res.e21y < threshold;
      bool is_projection_overlap = res.e11x < res.e21x and res.e22x < res.e12x;
      res.is_violation =
          is_outside_to_outside and is_too_close and is_projection_overlap;
    }
  }
}

void run_space_check(const polygon&   polygon1,
                     const polygon&   polygon2,
                     int              threshold,
                     coord*           coords1,
                     coord*           coords2,
                     check_result*    results,
                     check_result*    result_host,
                     hipGraphExec_t& graph_exec) {
  hipStream_t stream1 = nullptr;
  hipStream_t stream2 = nullptr;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  int         size1 = (polygon1.points.size() - 1) / 2;
  int         size2 = (polygon2.points.size() - 1) / 2;
  hipGraph_t graph;
  // Two streams will be launched:
  //   s
  //  / \  // to avoid -Wcomment due to '\'
  // s1 s2
  //  \ /
  //   e
  hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal);
  // construct cuda graph for width check here
  hipMemcpyAsync(coords1, polygon1.points.data(),
                  sizeof(coord) * polygon1.points.size(),
                  hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(coords2, polygon2.points.data(),
                  sizeof(coord) * polygon2.points.size(),
                  hipMemcpyHostToDevice, stream2);
  hipEvent_t e1;  // sync memcpy
  hipEventCreate(&e1);
  hipEventRecord(e1, stream1);
  hipStreamWaitEvent(stream2, e1);

  int num_parallel_checks = size1 * size2 / 4;
  space_check_kernel<<<(num_parallel_checks + 127) / 128, 128, 0, stream1>>>(
      coords1, coords2, size1, size2, 0, threshold, results);
  space_check_kernel<<<(num_parallel_checks + 127) / 128, 128, 0, stream2>>>(
      coords1, coords2, size1, size2, 1, threshold, results);

  hipEvent_t e2;  // sync all kernel launch
  hipEventCreate(&e2);
  hipEventRecord(e2, stream2);
  hipStreamWaitEvent(stream1, e2);
  hipMemcpyAsync(result_host, results,
                  sizeof(check_result) * num_parallel_checks * 2,
                  hipMemcpyDeviceToHost, stream1);
  hipStreamEndCapture(stream1, &graph);
  auto perror = hipGetLastError();
  if (perror != hipSuccess) {  // TODO: change to OpenDRC exception
    throw std::runtime_error("CUDA pre error: " + std::to_string(perror));
  }

  hipGraphExecUpdateResult update_result;
  // If we've already instantiated the graph, try to update it directly
  // and avoid the instantiation overhead
  if (graph_exec != nullptr) {
    hipGraphNode_t error_node;
    // If the graph fails to update, errorNode will be set to the
    // node causing the failure and updateResult will be set to a
    // reason code.
    hipGraphExecUpdate(graph_exec, graph, &error_node, &update_result);
  }

  // Instantiate during the first iteration or whenever the update
  // fails for any reason
  if (graph_exec == nullptr || update_result != hipGraphExecUpdateSuccess) {
    // If a previous update failed, destroy the hipGraphExec_t
    // before re-instantiating it
    if (graph_exec != nullptr) {
      hipGraphExecDestroy(graph_exec);
    }
    // Instantiate graphExec from graph. The error node and
    // error message parameters are unused here.
    hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0);
  }
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipGraphDestroy(graph);
  hipGraphLaunch(graph_exec, 0);
  hipDeviceSynchronize();
  auto error = hipGetLastError();
  if (error != hipSuccess) {  // TODO: change to OpenDRC exception
    throw std::runtime_error("CUDA error: " + std::to_string(error));
  }
}

__global__ void check_vertical(v_edge*       v1,
                               v_edge*       v2,
                               int           size1,
                               int           size2,
                               int           threshold,
                               check_result* results) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= size1 * size2) {
    return;
  }
  int p1 = tid / size2;
  int p2 = tid % size2;

  check_result& res = results[tid];

  res.e11x = v1[p1].x;
  res.e11y = v1[p1].y1;
  res.e12x = v1[p1].x;
  res.e12y = v1[p1].y2;
  res.e21x = v2[p2].x;
  res.e21y = v2[p2].y1;
  res.e22x = v2[p2].x;
  res.e22y = v2[p2].y2;

  if (res.e11x < res.e21x) {
    // e11 e22
    // e12 e21
    bool is_outside_to_outside = res.e11y > res.e12y and res.e21y < res.e22y;
    bool is_too_close          = res.e21x - res.e11x < threshold;
    bool is_projection_overlap = res.e11y < res.e21y and res.e22y < res.e12y;
    res.is_violation =
        is_outside_to_outside and is_too_close and is_projection_overlap;
    if (res.is_violation) {
      printf("T[%d]: (%d, %d), (%d, %d), (%d, %d), (%d, %d)\n", tid, res.e11x,
             res.e11y, res.e12x, res.e12y, res.e21x, res.e21y, res.e22x,
             res.e22y);
    }

  } else {
    // e21 e12
    // e22 e11
    bool is_outside_to_outside = res.e21y > res.e22y and res.e11y < res.e21y;
    bool is_too_close          = res.e11x - res.e21x < threshold;
    bool is_projection_overlap = res.e21y < res.e11y and res.e12y < res.e22y;
    res.is_violation =
        is_outside_to_outside and is_too_close and is_projection_overlap;
  }
}

__global__ void check_horizontal(h_edge*       h1,
                                 h_edge*       h2,
                                 int           size1,
                                 int           size2,
                                 int           threshold,
                                 check_result* results) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= size1 * size2) {
    return;
  }
  int p1 = tid / size2;
  int p2 = tid % size2;

  check_result& res = results[tid];

  res.e11x = h1[p1].x1;
  res.e11y = h1[p1].y;
  res.e12x = h1[p1].x2;
  res.e12y = h1[p1].y;
  res.e21x = h2[p2].x1;
  res.e21y = h2[p2].y;
  res.e22x = h2[p2].x2;
  res.e22y = h2[p2].y;

  if (res.e11y < res.e22y) {
    // e22 e21
    // e11 e12
    bool is_outside_to_outside = res.e11x < res.e12x and res.e21x > res.e22x;
    bool is_too_close          = res.e21y - res.e11y < threshold;
    bool is_projection_overlap = res.e21x < res.e11x and res.e12x < res.e22x;
    res.is_violation =
        is_outside_to_outside and is_too_close and is_projection_overlap;
  } else {
    // e12 e11
    // e21 e22
    bool is_outside_to_outside = res.e21x < res.e22x and res.e11x > res.e12x;
    bool is_too_close          = res.e11y - res.e21y < threshold;
    bool is_projection_overlap = res.e11x < res.e21x and res.e22x < res.e12x;
    res.is_violation =
        is_outside_to_outside and is_too_close and is_projection_overlap;
  }
}

void space_check(const odrc::core::database& db,
                 const int                   layer1,
                 const int                   layer2,
                 const int                   threshold) {
  hipStream_t    stream;
  hipGraphExec_t graph_exec = nullptr;
  hipError_t     error;
  coord*          coord_buffer1      = nullptr;
  coord*          coord_buffer2      = nullptr;
  check_result*   check_results      = nullptr;
  check_result*   check_results_host = nullptr;

  // TODO: remove magic numbers
  hipStreamCreate(&stream);
  hipMallocAsync((void**)&coord_buffer1, sizeof(coord) * 201, stream);
  hipMallocAsync((void**)&coord_buffer2, sizeof(coord) * 201, stream);
  hipMallocAsync((void**)&check_results, sizeof(check_result) * 20000, stream);
  hipHostMalloc((void**)&check_results_host, sizeof(check_result) * 20000);
  error = hipStreamSynchronize(stream);
  if (error != hipSuccess) {  // TODO: change to OpenDRC exception
    throw std::runtime_error("CUDA error: " + std::to_string(error));
  }
  hipStreamDestroy(stream);

  // result memoiozation
  std::unordered_map<std::string, bool> checked_results;

  // The structure to represent an inter-poly check.
  // It cares about layer1 for cell1 and layer2 for cell2.
  // Only the top-most cell is not cell_ref.
  struct _task {
    std::variant<const polygon*, const cell_ref*> object1;
    const cell_ref*                               object2;
    bool                                          is_every_subtask_done = false;
  };

  cell_ref top_cell_ref{db.cells.back().name, odrc::core::coord{0, 0}, {}};

  std::stack<_task> tasks;
  tasks.push(_task{&top_cell_ref, &top_cell_ref, false});

  while (not tasks.empty()) {
    std::cout << tasks.size() << std::endl;
    auto task = tasks.top();
    tasks.pop();
    const cell_ref** r1    = std::get_if<const cell_ref*>(&task.object1);
    const auto&      cell2 = db.get_cell(task.object2->cell_name);
    if (task.is_every_subtask_done) {  // just to see if memoization is possible
      if (r1 != nullptr and *r1 == task.object2) {
        checked_results.emplace((*r1)->cell_name, true);
      }
      continue;
    }
    if (r1 == nullptr) {  // object1 is polygon
      std::cout << "object1 is polygon" << std::endl;
      tasks.push(_task{task.object1, task.object2, true});
      const auto& polygon1 = *std::get<const polygon*>(task.object1);
      assert(polygon1.layer == layer1);  // should be enqueued otherwise

      // polygon vs polygon are sent to run_space_check directly
      std::cout << "polygon vs polygon are sent to run_space_check directly"
                << std::endl;

      for (const auto& polygon2 : cell2.polygons) {
        if (polygon2.layer == layer2 and polygon1.is_touching(polygon2)) {
          run_space_check(polygon1, polygon2, threshold, coord_buffer1,
                          coord_buffer2, check_results, check_results_host,
                          graph_exec);
        }
      }

      // polygon vs cell_ref are enqueued
      std::cout << "polygon vs cell_ref are enqueued" << std::endl;

      for (const auto& cell_ref2 : cell2.cell_refs) {
        const auto& the_cell = db.get_cell(cell_ref2.cell_name);
        if (the_cell.is_touching(layer2) and cell_ref2.is_touching(polygon1)) {
          tasks.push(_task{&polygon1, &cell_ref2, false});
        }
      }

    } else {  // object1 is cell
      std::cout << "object1 is cell" << std::endl;

      // NOTE: (a^M b^M) and (b^M a^M) duplicates
      // to fix, assign arbitrary id to cells to mark unique checks
      const auto& cell1 =
          db.get_cell(std::get<const cell_ref*>(task.object1)->cell_name);
      std::cout << "  cell name:" << cell1.name << std::endl;
      tasks.push(_task{task.object1, task.object2, true});
      for (const auto& polygon1 : cell1.polygons) {
        if (polygon1.layer != layer1) {
          continue;
        }
        // polygon vs polygon are sent to run_space_check directly
        std::cout << "polygon vs polygon are sent to run_space_check directly"
                  << std::endl;
        for (const auto& polygon2 : cell2.polygons) {
          if (polygon2.layer == layer2 and polygon1.is_touching(polygon2)) {
            run_space_check(polygon1, polygon2, threshold, coord_buffer1,
                            coord_buffer2, check_results, check_results_host,
                            graph_exec);
          }
        }
        // polygon vs cell_ref are enqueued
        std::cout << "polygon vs cell_ref are enqueued" << std::endl;
        for (const auto& cell_ref2 : cell2.cell_refs) {
          const auto& the_cell = db.get_cell(cell_ref2.cell_name);
          if (the_cell.is_touching(layer2) and
              cell_ref2.is_touching(polygon1)) {
            tasks.push(_task{&polygon1, &cell_ref2, false});
          }
        }
      }

      // cell_ref vs cell_ref are enqueued
      std::cout << "cell_ref vs cell_ref are enqueued" << std::endl;
      for (const auto& cell_ref1 : cell1.cell_refs) {
        const auto& the_cell1 = db.get_cell(cell_ref1.cell_name);
        if (the_cell1.is_touching(layer1)) {
          for (const auto& cell_ref2 : cell2.cell_refs) {
            const auto& the_cell2 = db.get_cell(cell_ref2.cell_name);
            if (the_cell2.is_touching(layer2) and
                cell_ref1.is_touching(cell_ref2)) {
              std::cout << the_cell1.name << " vs " << the_cell2.name
                        << std::endl;
              if (false and the_cell1.depth == 1 and the_cell2.depth == 1) {
                h_edge* h_edges1;
                v_edge* v_edges1;
                h_edge* h_edges2;
                v_edge* v_edges2;
                hipMalloc((void**)&h_edges1,
                           sizeof(h_edge) * cell_ref1.h_edges.size());
                hipMalloc((void**)&v_edges1,
                           sizeof(h_edge) * cell_ref1.v_edges.size());
                hipMalloc((void**)&h_edges2,
                           sizeof(h_edge) * cell_ref2.h_edges.size());
                hipMalloc((void**)&v_edges2,
                           sizeof(h_edge) * cell_ref2.v_edges.size());
                hipMemcpy(h_edges1, cell_ref1.h_edges.data(),
                           sizeof(h_edge) * cell_ref1.h_edges.size(),
                           hipMemcpyHostToDevice);
                hipMemcpy(v_edges1, cell_ref1.v_edges.data(),
                           sizeof(v_edge) * cell_ref1.h_edges.size(),
                           hipMemcpyHostToDevice);
                hipMemcpy(h_edges2, cell_ref2.h_edges.data(),
                           sizeof(h_edge) * cell_ref2.h_edges.size(),
                           hipMemcpyHostToDevice);
                hipMemcpy(v_edges2, cell_ref2.v_edges.data(),
                           sizeof(v_edge) * cell_ref2.h_edges.size(),
                           hipMemcpyHostToDevice);

                check_result* check_results_fast = nullptr;
                hipMalloc((void**)&check_results_fast,
                           sizeof(check_result) * cell_ref1.h_edges.size() *
                               cell_ref2.h_edges.size());
                check_result* check_results_fast_host = nullptr;
                hipHostMalloc((void**)&check_results_fast,
                               sizeof(check_result) * cell_ref1.h_edges.size() *
                                   cell_ref2.h_edges.size());
                int size1 = cell_ref1.h_edges.size();
                int size2 = cell_ref2.h_edges.size();
                check_horizontal<<<(size1 * size2 + 127) / 128, 128>>>(
                    h_edges1, h_edges2, cell_ref1.h_edges.size(),
                    cell_ref2.h_edges.size(), threshold, check_results);
                hipDeviceSynchronize();
                hipMemcpy(check_results_fast_host, check_results_fast,
                           sizeof(check_result) * size1 * size2,
                           hipMemcpyDeviceToHost);
                check_vertical<<<(size1 * size2 + 127) / 128, 128>>>(
                    v_edges1, v_edges2, cell_ref1.v_edges.size(),
                    cell_ref2.v_edges.size(), threshold, check_results);
                hipDeviceSynchronize();
                hipMemcpy(check_results_fast_host, check_results_fast,
                           sizeof(check_result) * size1 * size2,
                           hipMemcpyDeviceToHost);
              } else {
                tasks.push(_task{&cell_ref1, &cell_ref2, false});
              }
            }
          }
        }
      }
    }
  }
}
}  // namespace odrc